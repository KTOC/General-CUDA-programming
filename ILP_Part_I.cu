#include "hip/hip_runtime.h"
#include<stdio.h>

#define N_ITERATIONS 8192

#include "Utilities.cuh"
#include "TimingGPU.cuh"

#define BLOCKSIZE	512

//#define DEBUG

/********************************************************/
/* KERNEL0 - NO INSTRUCTION LEVEL PARALLELISM (ILP = 0) */
/********************************************************/
__global__ void kernel0(int * __restrict__ d_a, const int * __restrict__ d_b, const int * __restrict__ d_c, const int N) {

	const int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < N) {

		int a = d_a[tid];
		int b = d_b[tid];
		int c = d_c[tid];

		for (unsigned int i = 0; i < N_ITERATIONS; i++) {
			a = a * b + c;
		}

		d_a[tid] = a;
	}

}

/*****************************************************/
/* KERNEL1 - INSTRUCTION LEVEL PARALLELISM (ILP = 2) */
/*****************************************************/
__global__ void kernel1(int * __restrict__ d_a, const int * __restrict__ d_b, const int * __restrict__ d_c, const int N) {

	const int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < N / 2) {

		int a1 = d_a[tid];
		int b1 = d_b[tid];
		int c1 = d_c[tid];

		int a2 = d_a[tid + N / 2];
		int b2 = d_b[tid + N / 2];
		int c2 = d_c[tid + N / 2];

		for (unsigned int i = 0; i < N_ITERATIONS; i++) {
			a1 = a1 * b1 + c1;
			a2 = a2 * b2 + c2;
		}

		d_a[tid] = a1;
		d_a[tid + N / 2] = a2;
	}

}

/*****************************************************/
/* KERNEL2 - INSTRUCTION LEVEL PARALLELISM (ILP = 4) */
/*****************************************************/
__global__ void kernel2(int * __restrict__ d_a, const int * __restrict__ d_b, const int * __restrict__ d_c, const int N) {

	const int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < N / 4) {

		int a1 = d_a[tid];
		int b1 = d_b[tid];
		int c1 = d_c[tid];

		int a2 = d_a[tid + N / 4];
		int b2 = d_b[tid + N / 4];
		int c2 = d_c[tid + N / 4];

		int a3 = d_a[tid + N / 2];
		int b3 = d_b[tid + N / 2];
		int c3 = d_c[tid + N / 2];

		int a4 = d_a[tid + 3 * N / 4];
		int b4 = d_b[tid + 3 * N / 4];
		int c4 = d_c[tid + 3 * N / 4];

		for (unsigned int i = 0; i < N_ITERATIONS; i++) {
			a1 = a1 * b1 + c1;
			a2 = a2 * b2 + c2;
			a3 = a3 * b3 + c3;
			a4 = a4 * b4 + c4;
		}

		d_a[tid] = a1;
		d_a[tid + N / 4] = a2;
		d_a[tid + N / 2] = a3;
		d_a[tid + 3 * N / 4] = a4;
	}

}

/********/
/* MAIN */
/********/
int main() {

	const int N = 8192 * 4;

	TimingGPU timerGPU;

	int *h_a = (int*)malloc(N*sizeof(int));
	int *h_a_result_host = (int*)malloc(N*sizeof(int));
	int *h_a_result_device = (int*)malloc(N*sizeof(int));
	int *h_b = (int*)malloc(N*sizeof(int));
	int *h_c = (int*)malloc(N*sizeof(int));

	for (int i = 0; i<N; i++) {
		h_a[i] = 2;
		h_b[i] = 1;
		h_c[i] = 2;
		h_a_result_host[i] = h_a[i];
		for (unsigned int k = 0; k < N_ITERATIONS; k++) {
			h_a_result_host[i] = h_a_result_host[i] * h_b[i] + h_c[i];
		}
	}

	int *d_a; gpuErrchk(hipMalloc((void**)&d_a, N*sizeof(int)));
	int *d_b; gpuErrchk(hipMalloc((void**)&d_b, N*sizeof(int)));
	int *d_c; gpuErrchk(hipMalloc((void**)&d_c, N*sizeof(int)));

	gpuErrchk(hipMemcpy(d_a, h_a, N*sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_b, h_b, N*sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_c, h_c, N*sizeof(int), hipMemcpyHostToDevice));

	/***********/
	/* KERNEL0 */
	/***********/
	timerGPU.StartCounter();
	kernel0 << <iDivUp(N, BLOCKSIZE), BLOCKSIZE >> >(d_a, d_b, d_c, N);
#ifdef DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif
	// --- Remember: timing is in ms
	printf("GFlops = %f\n", (1.e-6)*((float)N*(float)N_ITERATIONS) / timerGPU.GetCounter());
	gpuErrchk(hipMemcpy(h_a_result_device, d_a, N*sizeof(int), hipMemcpyDeviceToHost));
	for (int i = 0; i<N; i++) if (h_a_result_device[i] != h_a_result_host[i]) { printf("Error at i=%i! Host = %i; Device = %i\n", i, h_a_result_host[i], h_a_result_device[i]); return 1; }

	/***********/
	/* KERNEL1 */
	/***********/
	gpuErrchk(hipMemcpy(d_a, h_a, N*sizeof(int), hipMemcpyHostToDevice));
	timerGPU.StartCounter();
	kernel1 << <iDivUp(N / 2, BLOCKSIZE), BLOCKSIZE >> >(d_a, d_b, d_c, N);
#ifdef DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif
	// --- Remember: timing is in ms
	printf("GFlops = %f\n", (1.e-6)*((float)N*(float)N_ITERATIONS) / timerGPU.GetCounter());
	gpuErrchk(hipMemcpy(h_a_result_device, d_a, N*sizeof(int), hipMemcpyDeviceToHost));
	for (int i = 0; i<N; i++) if (h_a_result_device[i] != h_a_result_host[i]) { printf("Error at i=%i! Host = %i; Device = %i\n", i, h_a_result_host[i], h_a_result_device[i]); return 1; }

	/***********/
	/* KERNEL2 */
	/***********/
	gpuErrchk(hipMemcpy(d_a, h_a, N*sizeof(int), hipMemcpyHostToDevice));
	timerGPU.StartCounter();
	kernel2 << <iDivUp(N / 4, BLOCKSIZE), BLOCKSIZE >> >(d_a, d_b, d_c, N);
#ifdef DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif
	// --- Remember: timing is in ms
	printf("GFlops = %f\n", (1.e-6)*(float)((float)N*(float)N_ITERATIONS) / timerGPU.GetCounter());
	gpuErrchk(hipMemcpy(h_a_result_device, d_a, N*sizeof(int), hipMemcpyDeviceToHost));
	for (int i = 0; i<N; i++) if (h_a_result_device[i] != h_a_result_host[i]) { printf("Error at i=%i! Host = %i; Device = %i\n", i, h_a_result_host[i], h_a_result_device[i]); return 1; }

	return 0;

}
