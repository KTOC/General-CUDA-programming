#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<>

#include "Utilities.cuh"
#include "TimingGPU.cuh"

#define BLOCKSIZE_x 16
#define BLOCKSIZE_y 16

/******************/
/* TEST KERNEL 2D */
/******************/
__global__ void test_kernel_2D(float * __restrict__ devPtrA, float * __restrict__ devPtrB, float * __restrict__ devPtrC, const int Nrows, const int Ncols)
{
	int    tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int    tidy = blockIdx.y * blockDim.y + threadIdx.y;

	if ((tidx < Ncols) && (tidy < Nrows))
		devPtrA[tidy * Ncols + tidx] = devPtrA[tidy * Ncols + tidx] + devPtrB[tidy * Ncols + tidx] + devPtrC[tidy * Ncols + tidx];
}

/**************************/
/* TEST KERNEL PITCHED 2D */
/**************************/
__global__ void test_kernel_Pitched_2D(float * __restrict__ devPtrA, float * __restrict__ devPtrB, float * __restrict__ devPtrC, const size_t pitchA, const size_t pitchB, const size_t pitchC, const int Nrows, const int Ncols)
{
	int    tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int    tidy = blockIdx.y * blockDim.y + threadIdx.y;

	if ((tidx < Ncols) && (tidy < Nrows))
	{
		float *row_a = (float *)((char*)devPtrA + tidy * pitchA);
		float *row_b = (float *)((char*)devPtrB + tidy * pitchB);
		float *row_c = (float *)((char*)devPtrC + tidy * pitchC);
		row_a[tidx] = row_a[tidx] + row_b[tidx] + row_c[tidx];
	}
}

/********/
/* MAIN */
/********/
int main()
{
	const int Nrows = 7100;
	const int Ncols = 2300;

	TimingGPU timerGPU;

	float *hostPtrA = (float *)malloc(Nrows * Ncols * sizeof(float));
	float *hostPtrB = (float *)malloc(Nrows * Ncols * sizeof(float));
	float *hostPtrC = (float *)malloc(Nrows * Ncols * sizeof(float));
	float *devPtrA, *devPtrPitchedA;
	float *devPtrB, *devPtrPitchedB;
	float *devPtrC, *devPtrPitchedC;
	size_t pitchA, pitchB, pitchC;

	for (int i = 0; i < Nrows; i++)
		for (int j = 0; j < Ncols; j++) {
		hostPtrA[i * Ncols + j] = 1.f;
		hostPtrB[i * Ncols + j] = 2.f;
		hostPtrC[i * Ncols + j] = 3.f;
		//printf("row %i column %i value %f \n", i, j, hostPtr[i][j]);
		}

	// --- 2D non-pitched allocation and host->device memcopy
	gpuErrchk(hipMalloc(&devPtrA, Nrows * Ncols * sizeof(float)));
	gpuErrchk(hipMalloc(&devPtrB, Nrows * Ncols * sizeof(float)));
	gpuErrchk(hipMalloc(&devPtrC, Nrows * Ncols * sizeof(float)));
	gpuErrchk(hipMemcpy(devPtrA, hostPtrA, Nrows * Ncols * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devPtrB, hostPtrB, Nrows * Ncols * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devPtrC, hostPtrC, Nrows * Ncols * sizeof(float), hipMemcpyHostToDevice));

	// --- 2D pitched allocation and host->device memcopy
	gpuErrchk(hipMallocPitch(&devPtrPitchedA, &pitchA, Ncols * sizeof(float), Nrows));
	gpuErrchk(hipMallocPitch(&devPtrPitchedB, &pitchB, Ncols * sizeof(float), Nrows));
	gpuErrchk(hipMallocPitch(&devPtrPitchedC, &pitchC, Ncols * sizeof(float), Nrows));
	gpuErrchk(hipMemcpy2D(devPtrPitchedA, pitchA, hostPtrA, Ncols * sizeof(float), Ncols*sizeof(float), Nrows, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy2D(devPtrPitchedB, pitchB, hostPtrB, Ncols * sizeof(float), Ncols*sizeof(float), Nrows, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy2D(devPtrPitchedC, pitchC, hostPtrC, Ncols * sizeof(float), Ncols*sizeof(float), Nrows, hipMemcpyHostToDevice));

	dim3 gridSize(iDivUp(Ncols, BLOCKSIZE_x), iDivUp(Nrows, BLOCKSIZE_y));
	dim3 blockSize(BLOCKSIZE_y, BLOCKSIZE_x);

	timerGPU.StartCounter();
	test_kernel_2D << <gridSize, blockSize >> >(devPtrA, devPtrB, devPtrC, Nrows, Ncols);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
	printf("Non-pitched - Time = %f; Memory = %i bytes \n", timerGPU.GetCounter(), Nrows * Ncols * sizeof(float));

	timerGPU.StartCounter();
	test_kernel_Pitched_2D << <gridSize, blockSize >> >(devPtrPitchedA, devPtrPitchedB, devPtrPitchedC, pitchA, pitchB, pitchC, Nrows, Ncols);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
	printf("Pitched - Time = %f; Memory = %i bytes \n", timerGPU.GetCounter(), Nrows * pitchA);

	//gpuErrchk(hipMemcpy2D(hostPtr, Ncols * sizeof(float), devPtrPitched, pitch, Ncols * sizeof(float), Nrows, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(hostPtrA, devPtrA, Nrows * Ncols * sizeof(float), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(hostPtrB, devPtrB, Nrows * Ncols * sizeof(float), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(hostPtrC, devPtrC, Nrows * Ncols * sizeof(float), hipMemcpyDeviceToHost));

	//for (int i = 0; i < Nrows; i++) 
	//	for (int j = 0; j < Ncols; j++) 
	//		printf("row %i column %i value %f \n", i, j, hostPtr[i * Ncols + j]);

	return 0;

}
