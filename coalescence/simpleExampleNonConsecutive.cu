#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCKSIZE 128

/*******************/
/* iDivUp FUNCTION */
/*******************/
int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }

/********************/
/* CUDA ERROR CHECK */
/********************/
// --- Credit to http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) { exit(code); }
	}
}

void gpuErrchk(hipError_t ans) { gpuAssert((ans), __FILE__, __LINE__); }

/*******************/
/* addCPU FUNCTION */
/*******************/
void addCPU(int *h_a, int *h_b, int *h_c, int N) {

	int h_a_tmp, h_b_tmp;

	for (int k = 0; k < N; k++) {

		if ((k % 2) == 0) {
			h_a_tmp = h_a[k + 1];
			h_b_tmp = h_b[k + 1];
		}
		else {
			h_a_tmp = h_a[k - 1];
			h_b_tmp = h_b[k - 1];
		}

		h_c[k] = h_a_tmp + h_b_tmp;
	}

}

/***************************************************/
/* addGPU FUNCTION WITH NON-CONSECUTIVE LOAD/STORE */
/***************************************************/
__global__ void addGPUNonConsecutive(int *d_a, int *d_b, int *d_c, int N) {

	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid >= N) return;

	int d_a_tmp, d_b_tmp;

	if ((tid % 2) == 0) {
		d_a_tmp = d_a[tid + 1];
		d_b_tmp = d_b[tid + 1];
	}
	else {
		d_a_tmp = d_a[tid - 1];
		d_b_tmp = d_b[tid - 1];
	}

	d_c[tid] = d_a_tmp + d_b_tmp;

}

/********/
/* MAIN */
/********/
int main() {

	const int N = 256;

	// --- Allocating host memory for data and results
	int *h_a = (int *)malloc(N * sizeof(int));
	int *h_b = (int *)malloc(N * sizeof(int));
	int *h_c = (int *)malloc(N * sizeof(int));
	int *h_c_device = (int *)malloc(N * sizeof(int));

	// --- Allocating device memory for data and results
	int *d_a, *d_b, *d_c;
	gpuErrchk(hipMalloc(&d_a, N * sizeof(int)));
	gpuErrchk(hipMalloc(&d_b, N * sizeof(int)));
	gpuErrchk(hipMalloc(&d_c, N * sizeof(int)));

	// --- Filling the input vectors on host memory
	for (int k = 0; k < N; k++) {
		h_a[k] = k;
		h_b[k] = 2 * k;
	}

	// --- Moving data from host to device
	gpuErrchk(hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice));

	addCPU(h_a, h_b, h_c, N);

	//addGPU << <iDivUp(N, BLOCKSIZE), BLOCKSIZE >> >(d_a, d_b, d_c, N);
	addGPUNonConsecutive << <iDivUp(N, BLOCKSIZE), BLOCKSIZE >> >(d_a, d_b, d_c, N);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipPeekAtLastError());

	gpuErrchk(hipMemcpy(h_c_device, d_c, N * sizeof(int), hipMemcpyDeviceToHost));

	for (int k = 0; k < N; k++) 
		if (h_c_device[k] != h_c[k]) {
			printf("Host and device results do not match for k = %d: h_c[%d] = %d; h_c_device[%d] = %d\n", k, k, h_c[k], k, h_c_device[k]);
		}

	printf("No errors found.\n");

	return 0;
}