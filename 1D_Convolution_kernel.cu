#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "TimingGPU.cuh"
#include "Utilities.cuh"

#define RG          10
#define BLOCKSIZE   8

/****************/
/* CPU FUNCTION */
/****************/
void h_convolution_1D(const float * __restrict__ h_Signal, const float * __restrict__ h_ConvKernel, float * __restrict__ h_Result_CPU, 
                      const int N, const int K) {

    for (int i = 0; i < N; i++) {

        float temp = 0.f;

        int N_start_point = i - (K / 2);

        for (int j = 0; j < K; j++) if (N_start_point + j >= 0 && N_start_point + j < N) {
            temp += h_Signal[N_start_point+ j] * h_ConvKernel[j];
        }

        h_Result_CPU[i] = temp;
    }
}

/********************/
/* BASIC GPU KERNEL */
/********************/
__global__ void d_convolution_1D_basic(const float * __restrict__ d_Signal, const float * __restrict__ d_ConvKernel, float * __restrict__ d_Result_GPU, 
                                       const int N, const int K) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    float temp = 0.f;

    int N_start_point = i - (K / 2);

    for (int j = 0; j < K; j++) if (N_start_point + j >= 0 && N_start_point + j < N) {
        temp += d_Signal[N_start_point+ j] * d_ConvKernel[j];
    }

    d_Result_GPU[i] = temp;
}

/***************************/
/* GPU KERNEL WITH CACHING */
/***************************/
__global__ void d_convolution_1D_caching(const float * __restrict__ d_Signal, const float * __restrict__ d_ConvKernel, float * __restrict__ d_Result_GPU, 
                                         const int N, const int K) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float d_Tile[BLOCKSIZE];

    d_Tile[threadIdx.x] = d_Signal[i];
    __syncthreads();

    float temp = 0.f;

    int N_start_point = i - (K / 2);

    for (int j = 0; j < K; j++) if (N_start_point + j >= 0 && N_start_point + j < N) {

            if ((N_start_point + j >= blockIdx.x * blockDim.x) && (N_start_point + j < (blockIdx.x + 1) * blockDim.x))

                // --- The signal element is in the tile loaded in the shared memory
                temp += d_Tile[threadIdx.x + j - (K / 2)] * d_ConvKernel[j]; 

            else

                // --- The signal element is not in the tile loaded in the shared memory
                temp += d_Signal[N_start_point + j] * d_ConvKernel[j];

    }

    d_Result_GPU[i] = temp;
}

/********/
/* MAIN */
/********/
int main(){

    const int N = 15;           // --- Signal length
    const int K = 5;            // --- Convolution kernel length

    float *h_Signal         = (float *)malloc(N * sizeof(float));
    float *h_Result_CPU     = (float *)malloc(N * sizeof(float));
    float *h_Result_GPU     = (float *)malloc(N * sizeof(float));
    float *h_ConvKernel     = (float *)malloc(K * sizeof(float));

    float *d_Signal;        gpuErrchk(hipMalloc(&d_Signal,     N * sizeof(float)));
    float *d_Result_GPU;    gpuErrchk(hipMalloc(&d_Result_GPU, N * sizeof(float)));
    float *d_ConvKernel;    gpuErrchk(hipMalloc(&d_ConvKernel, K * sizeof(float)));

    for (int i=0; i < N; i++) { h_Signal[i] = (float)(rand() % RG); }

    for (int i=0; i < K; i++) { h_ConvKernel[i] = (float)(rand() % RG); }

    gpuErrchk(hipMemcpy(d_Signal,      h_Signal,       N * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_ConvKernel,  h_ConvKernel,   K * sizeof(float), hipMemcpyHostToDevice));

    h_convolution_1D(h_Signal, h_ConvKernel, h_Result_CPU, N, K);

    d_convolution_1D_basic<<<iDivUp(N, BLOCKSIZE), BLOCKSIZE>>>(d_Signal, d_ConvKernel, d_Result_GPU, N, K);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(h_Result_GPU, d_Result_GPU, N * sizeof(float), hipMemcpyDeviceToHost));

    for (int i = 0; i < N; i++) if (h_Result_CPU[i] != h_Result_GPU[i]) {printf("mismatch2 at %d, cpu: %d, gpu %d\n", i, h_Result_CPU[i], h_Result_GPU[i]); return 1;}

    printf("Test basic passed\n");

    d_convolution_1D_caching<<<iDivUp(N, BLOCKSIZE), BLOCKSIZE>>>(d_Signal, d_ConvKernel, d_Result_GPU, N, K);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(h_Result_GPU, d_Result_GPU, N * sizeof(float), hipMemcpyDeviceToHost));

    for (int i = 0; i < N; i++) if (h_Result_CPU[i] != h_Result_GPU[i]) {printf("mismatch2 at %d, cpu: %d, gpu %d\n", i, h_Result_CPU[i], h_Result_GPU[i]); return 1;}

    printf("Test caching passed\n");

    return 0;
}
