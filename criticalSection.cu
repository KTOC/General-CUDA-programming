#include "hip/hip_runtime.h"
#include <stdio.h>

#include "Utilities.cuh"

#define NUMBLOCKS  512
#define NUMTHREADS 512 * 2

/***************/
/* LOCK STRUCT */
/***************/
struct Lock {

	int *d_state;

	// --- Constructor
	Lock(void) {
		int h_state = 0;										// --- Host side lock state initializer
		gpuErrchk(hipMalloc((void **)&d_state, sizeof(int)));	// --- Allocate device side lock state
		gpuErrchk(hipMemcpy(d_state, &h_state, sizeof(int), hipMemcpyHostToDevice)); // --- Initialize device side lock state
	}

	// --- Destructor
	__host__ __device__ ~Lock(void) { 
#if !defined(__HIPCC__)
		gpuErrchk(hipFree(d_state)); 
#else

#endif	
	}

	// --- Lock function
	__device__ void lock(void) { while (atomicCAS(d_state, 0, 1) != 0); }

	// --- Unlock function
	__device__ void unlock(void) { atomicExch(d_state, 0); }
};

/*************************************/
/* BLOCK COUNTER KERNEL WITHOUT LOCK */
/*************************************/
__global__ void blockCounterUnlocked(int *nblocks) {
	
	if (threadIdx.x == 0) {	*nblocks = *nblocks + 1; }
}

/**********************************/
/* BLOCK COUNTER KERNEL WITH LOCK */
/**********************************/
__global__ void blockCounterLocked(Lock lock, int *nblocks) {

	if (threadIdx.x == 0) {
		lock.lock();
		*nblocks = *nblocks + 1;
		lock.unlock();
	}
}

/****************************************/
/* BLOCK COUNTER KERNEL WITH WRONG LOCK */
/****************************************/
__global__ void blockCounter2(Lock lock, int *nblocks) {
	
	lock.lock();
	if (threadIdx.x == 0) { *nblocks = *nblocks + 1; }
	lock.unlock();
}

/********/
/* MAIN */
/********/
int main(){

	int h_counting, *d_counting;
	Lock lock;

	gpuErrchk(hipMalloc(&d_counting, sizeof(int)));

	// --- Unlocked case
	h_counting = 0;
	gpuErrchk(hipMemcpy(d_counting, &h_counting, sizeof(int), hipMemcpyHostToDevice));

	blockCounterUnlocked << <NUMBLOCKS, NUMTHREADS >> >(d_counting);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	gpuErrchk(hipMemcpy(&h_counting, d_counting, sizeof(int), hipMemcpyDeviceToHost));
	printf("Counting in the unlocked case: %i\n", h_counting);

	// --- Locked case
	h_counting = 0;
	gpuErrchk(hipMemcpy(d_counting, &h_counting, sizeof(int), hipMemcpyHostToDevice));

	blockCounterLocked << <NUMBLOCKS, NUMTHREADS >> >(lock, d_counting);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	gpuErrchk(hipMemcpy(&h_counting, d_counting, sizeof(int), hipMemcpyDeviceToHost));
	printf("Counting in the locked case: %i\n", h_counting);

	gpuErrchk(hipFree(d_counting));
}
