#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>

#define RADIUS        3
#define BLOCK_SIZE_X  8
#define BLOCK_SIZE_Y  8

/*******************/
/* iDivUp FUNCTION */
/*******************/
int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/**********/
/* KERNEL */
/**********/
__global__ void moving_average(unsigned int *in, unsigned int *out, unsigned int M, unsigned int N) {

    __shared__ unsigned int temp[BLOCK_SIZE_Y][BLOCK_SIZE_X + 2 * RADIUS];

    unsigned int gindexx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int gindexy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int gindex  = gindexy * N + gindexx;

    unsigned int lindexx = threadIdx.x + RADIUS;
    unsigned int lindexy = threadIdx.y;

    // --- Read input elements into shared memory
    temp[lindexy][lindexx] = ((gindexx < N)&&(gindexy < M))? in[gindex] : 0;
    if (threadIdx.x < RADIUS) {
        temp[lindexy][threadIdx.x] = ((gindexx >= RADIUS)&&(gindexx < (N + RADIUS))&&(gindexy < M)) ? in[gindex - RADIUS] : 0;
        temp[lindexy][threadIdx.x + (RADIUS + min(BLOCK_SIZE_X, N - blockIdx.x * BLOCK_SIZE_X))] = (((gindexx + min(BLOCK_SIZE_X, N - blockIdx.x * BLOCK_SIZE_X)) < N)&&(gindexy < M))? in[gindexy * N + gindexx + min(BLOCK_SIZE_X, N - blockIdx.x * BLOCK_SIZE_X)] : 0;
        if ((threadIdx.y == 0)&&(gindexy < M)&&((gindexx + BLOCK_SIZE_X) < N)&&(gindexy < M)) printf("Inside 2 - tidx = %i; bidx = %i; tidy = %i; bidy = %i; lindexx = %i; temp = %i\n", threadIdx.x, blockIdx.x, threadIdx.y, blockIdx.y, threadIdx.x + (RADIUS + BLOCK_SIZE_X), temp[lindexy][threadIdx.x + (RADIUS + BLOCK_SIZE_X)]);
    }

    __syncthreads();

    // --- Apply the stencil
    unsigned int result = 0;
    for (int offset = -RADIUS ; offset <= RADIUS ; offset++) {
        result += temp[lindexy][lindexx + offset];
    }

    // --- Store the result
    out[gindexy * N + gindexx] = result;
}

/********/
/* MAIN */
/********/
int main() {

    const unsigned int M        = 2;
    const unsigned int N        = 4 + 2 * RADIUS;

    const unsigned int constant = 3;

    thrust::device_vector<unsigned int> d_in(M * N, constant);
    thrust::device_vector<unsigned int> d_out(M * N);

    dim3 GridSize(iDivUp(N, BLOCK_SIZE_X), iDivUp(M, BLOCK_SIZE_Y));
    dim3 BlockSize(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    moving_average<<<GridSize, BlockSize>>>(thrust::raw_pointer_cast(d_in.data()), thrust::raw_pointer_cast(d_out.data()), M, N);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    thrust::host_vector<unsigned int> h_out = d_out;

    for (int j=0; j<M; j++) {
        for (int i=0; i<N; i++)
            printf("Element j = %i; i = %i; h_out = %i\n", j, i, h_out[N*j+i]);
    }

    return 0;

}
