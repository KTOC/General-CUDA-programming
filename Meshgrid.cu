#include "hip/hip_runtime.h"
#include <cstdio>

#include <thrust/pair.h>

#include "Matlab_like.cuh"
#include "Utilities.cuh"

#define BLOCKSIZE_MESHGRID_X	16
#define BLOCKSIZE_MESHGRID_Y	16

#define DEBUG

/*******************/
/* MESHGRID KERNEL */
/*******************/
template <class T>
__global__ void meshgrid_kernel(const T * __restrict__ x, size_t Nx, const float * __restrict__ y, size_t Ny, T * __restrict__ X, T * __restrict__ Y) 
{
	unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int tidy = blockIdx.y * blockDim.y + threadIdx.y;

	if ((tidx < Nx) && (tidy < Ny)) {	
		X[tidy * Nx + tidx] = x[tidx];
		Y[tidy * Nx + tidx] = y[tidy];
	}
}

/************/
/* MESHGRID */
/************/
template <class T>
thrust::pair<T *,T *> meshgrid(const T *x, const unsigned int Nx, const T *y, const unsigned int Ny) {
	
	T *X; gpuErrchk(hipMalloc((void**)&X, Nx * Ny * sizeof(T)));
	T *Y; gpuErrchk(hipMalloc((void**)&Y, Nx * Ny * sizeof(T)));

	dim3 BlockSize(BLOCKSIZE_MESHGRID_X, BLOCKSIZE_MESHGRID_Y);
	dim3 GridSize (iDivUp(Nx, BLOCKSIZE_MESHGRID_X), iDivUp(BLOCKSIZE_MESHGRID_Y, BLOCKSIZE_MESHGRID_Y));
	
	meshgrid_kernel<<<GridSize, BlockSize>>>(x, Nx, y, Ny, X, Y);
#ifdef DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

	return thrust::make_pair(X, Y);
}

/********/
/* MAIN */
/********/
int main()
{
	const int Nx = 3;
	const int Ny = 4;

	float *h_x = (float *)malloc(Nx * sizeof(float));
	float *h_y = (float *)malloc(Ny * sizeof(float));

	float *h_X = (float *)malloc(Nx * Ny * sizeof(float));
	float *h_Y = (float *)malloc(Nx * Ny * sizeof(float));

	for (int i = 0; i < Nx; i++) h_x[i] = i;
	for (int i = 0; i < Ny; i++) h_y[i] = i + 4.f;
	
	float *d_x;	gpuErrchk(hipMalloc(&d_x, Nx * sizeof(float)));
	float *d_y;	gpuErrchk(hipMalloc(&d_y, Ny * sizeof(float)));

	gpuErrchk(hipMemcpy(d_x, h_x, Nx * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_y, h_y, Ny * sizeof(float), hipMemcpyHostToDevice));
	
	thrust::pair<float *, float *> meshgrid_pointers = meshgrid(d_x, Nx, d_y, Ny);
	float *d_X = (float *)meshgrid_pointers.first;
	float *d_Y = (float *)meshgrid_pointers.second;

	gpuErrchk(hipMemcpy(h_X, d_X, Nx * Ny * sizeof(float), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_Y, d_Y, Nx * Ny * sizeof(float), hipMemcpyDeviceToHost));
 
	for (int j = 0; j < Ny; j++) {
		for (int i = 0; i < Nx; i++) {
			printf("i = %i; j = %i; x = %f; y = %f\n", i, j, h_X[j * Nx + i], h_Y[j * Nx + i]);
		}
	}

	return 0;

}
