#include "hip/hip_runtime.h"
#include <stdio.h>

texture<float, 1, hipReadModeElementType> texture_clamp;
texture<float, 1, hipReadModeElementType> texture_border;
texture<float, 1, hipReadModeElementType> texture_wrap;
texture<float, 1, hipReadModeElementType> texture_mirror;

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/******************************/
/* CUDA ADDRESS MODE CLAMPING */
/******************************/
__global__ void Test_texture_clamping(const int M) {

    printf("Texture clamping - i = %i; value = %f\n", -threadIdx.x, tex1D(texture_clamp, -(float)threadIdx.x));
    printf("Texture clamping - i = %i; value = %f\n", M + threadIdx.x, tex1D(texture_clamp, (float)(M + threadIdx.x)));

}

/****************************/
/* CUDA ADDRESS MODE BORDER */
/****************************/
__global__ void Test_texture_border(const int M) {

    printf("Texture border - i = %i; value = %f\n", -threadIdx.x, tex1D(texture_border, -(float)threadIdx.x));
    printf("Texture border - i = %i; value = %f\n", M + threadIdx.x, tex1D(texture_border, (float)(M + threadIdx.x)));

}

/**************************/
/* CUDA ADDRESS MODE WRAP */
/**************************/
__global__ void Test_texture_wrap(const int M) {

    printf("Texture wrap - i = %i; value = %f\n", -threadIdx.x, tex1D(texture_wrap, -(float)threadIdx.x/(float)M));
    printf("Texture wrap - i = %i; value = %f\n", M + threadIdx.x, tex1D(texture_wrap, (float)(M + threadIdx.x)/(float)M));

}

/****************************/
/* CUDA ADDRESS MODE MIRROR */
/****************************/
__global__ void Test_texture_mirror(const int M) {

    printf("Texture mirror - i = %i; value = %f\n", -threadIdx.x, tex1D(texture_mirror, -(float)threadIdx.x/(float)M));
    printf("Texture mirror - i = %i; value = %f\n", M + threadIdx.x, tex1D(texture_mirror, (float)(M + threadIdx.x)/(float)M));

}

/********/
/* MAIN */
/********/
void main(){

    const int M = 4;

    // --- Host side memory allocation and initialization
    float *h_data = (float*)malloc(M * sizeof(float));

    for (int i=0; i<M; i++) h_data[i] = (float)i;

    // --- Texture clamping
    hipArray* d_data_clamping = NULL; gpuErrchk(hipMallocArray(&d_data_clamping, &texture_clamp.channelDesc, M, 1)); 
    gpuErrchk(hipMemcpyToArray(d_data_clamping, 0, 0, h_data, M * sizeof(float), hipMemcpyHostToDevice)); 
    hipBindTextureToArray(texture_clamp, d_data_clamping); 
    texture_clamp.normalized = false; 
    texture_clamp.addressMode[0] = hipAddressModeClamp;

    dim3 dimBlock(2 * M, 1); dim3 dimGrid(1, 1);
    Test_texture_clamping<<<dimGrid,dimBlock>>>(M);

    printf("\n\n\n");

    // --- Texture border
    hipArray* d_data_border = NULL; gpuErrchk(hipMallocArray(&d_data_border, &texture_border.channelDesc, M, 1)); 
    gpuErrchk(hipMemcpyToArray(d_data_border, 0, 0, h_data, M * sizeof(float), hipMemcpyHostToDevice)); 
    hipBindTextureToArray(texture_border, d_data_border); 
    texture_border.normalized = false; 
    texture_border.addressMode[0] = hipAddressModeBorder;

    Test_texture_border<<<dimGrid,dimBlock>>>(M);

    printf("\n\n\n");

    // --- Texture wrap
    hipArray* d_data_wrap = NULL; gpuErrchk(hipMallocArray(&d_data_wrap, &texture_wrap.channelDesc, M, 1)); 
    gpuErrchk(hipMemcpyToArray(d_data_wrap, 0, 0, h_data, M * sizeof(float), hipMemcpyHostToDevice)); 
    hipBindTextureToArray(texture_wrap, d_data_wrap); 
    texture_wrap.normalized = true; 
    texture_wrap.addressMode[0] = hipAddressModeWrap;

    Test_texture_wrap<<<dimGrid,dimBlock>>>(M);

    printf("\n\n\n");

    // --- Texture mirror
    hipArray* d_data_mirror = NULL; gpuErrchk(hipMallocArray(&d_data_mirror, &texture_mirror.channelDesc, M, 1)); 
    gpuErrchk(hipMemcpyToArray(d_data_mirror, 0, 0, h_data, M * sizeof(float), hipMemcpyHostToDevice)); 
    hipBindTextureToArray(texture_mirror, d_data_mirror); 
    texture_mirror.normalized = true ; 
    texture_mirror.addressMode[0] = hipAddressModeMirror;

    Test_texture_mirror<<<dimGrid,dimBlock>>>(M);

    printf("\n\n\n");
}
